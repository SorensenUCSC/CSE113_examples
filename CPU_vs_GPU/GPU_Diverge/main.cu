
#include <hip/hip_runtime.h>
#include <thread>
#include <assert.h>
#include <chrono>
#include <iostream>
using namespace std::chrono;
using namespace std;

#define SIZE (1024*1024*32)
#define OUTER (128)

__global__ void vector_add(float * d_a, float * d_b, float * d_c, int size) {
  int global_id = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (global_id % 4 == 0) {
    for (int j = 0; j < OUTER; j++) {
      d_a[global_id] = d_b[global_id] + d_c[global_id];
    }
  }
  else if (global_id % 4 == 1) {
    for (int j = 0; j < OUTER; j++) {
      d_a[global_id] = d_b[global_id] * d_c[global_id];
    }    
  }
  else if (global_id % 4 == 2) {
    for (int j = 0; j < OUTER; j++) {
      d_a[global_id] = d_b[global_id] / d_c[global_id];
    }
  }
  else if (global_id % 4 == 3) {
    for (int j = 0; j < OUTER; j++) {
      d_a[global_id] = d_b[global_id] - d_c[global_id];
    }    
  }
}

__global__ void vector_add_chunked(float * d_a, float * d_b, float * d_c, int size) {
  int global_id = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (global_id < (size/4)*1) {
    for (int j = 0; j < OUTER; j++) {
      d_a[global_id] = d_b[global_id] + d_c[global_id];
    }
  }
  else if (global_id < (size/4)*2) {
    for (int j = 0; j < OUTER; j++) {
      d_a[global_id] = d_b[global_id] * d_c[global_id];
    }    
  }
  else if (global_id < (size/4)*3) {
    for (int j = 0; j < OUTER; j++) {
      d_a[global_id] = d_b[global_id] / d_c[global_id];
    }
  }
  else if (global_id < (size/4)*4) {
    for (int j = 0; j < OUTER; j++) {
      d_a[global_id] = d_b[global_id] - d_c[global_id];
    }    
  }
}



int main() {

  float *a = new float[SIZE];
  float *b = new float[SIZE];
  float *c = new float[SIZE];

  for(int i = 0; i < SIZE; i++) {
    a[i] = 0.0f;
    b[i] = i;
    c[i] = 1.0f;
  }

  float *d_a, *d_b, *d_c;
  int e = 0;
  e = hipMalloc(&d_a, SIZE*sizeof(float));
  e |= hipMalloc(&d_b, SIZE*sizeof(float));
  e |= hipMalloc(&d_c, SIZE*sizeof(float));




  e |= hipMemcpy(d_a, a, SIZE*sizeof(float), hipMemcpyHostToDevice);
  e |= hipMemcpy(d_b, b, SIZE*sizeof(float), hipMemcpyHostToDevice);
  e |= hipMemcpy(d_c, c, SIZE*sizeof(float), hipMemcpyHostToDevice);



  auto time_start = high_resolution_clock::now();
  vector_add<<<1024*32,1024>>>(d_a, d_b, d_c, SIZE);
  e |= hipDeviceSynchronize();

  auto time_end = high_resolution_clock::now();
  auto time_duration = duration_cast<nanoseconds>(time_end - time_start);
  double time_seconds = time_duration.count()/1000000000.0;
  
  cout << "timings: " << time_seconds << endl;

  e |= hipMemcpy(a, d_a, SIZE*sizeof(float), hipMemcpyDeviceToHost); 
      
  for (int i = 0; i < SIZE; i++) {
    assert(a[i] == (float(i) + 1.0f));    
  }  

  return 0;
}
