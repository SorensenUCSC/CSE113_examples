
#include <hip/hip_runtime.h>
#include <thread>
#include <assert.h>
#include <chrono>
#include <iostream>
using namespace std::chrono;
using namespace std;

#define SIZE (1024*1024)
#define OUTER (1024)

__global__ void vector_add(double * d_a, double * d_b, double * d_c, int size) {
  int global_id = (blockIdx.x * blockDim.x) + threadIdx.x;
  int i = global_id;
  for (int j = 0; j < OUTER; j++) {
    d_a[i] = d_b[i] + d_c[i];
  }
}


int main() {

  double *a = new double[SIZE];
  double *b = new double[SIZE];
  double *c = new double[SIZE];

  for(int i = 0; i < SIZE; i++) {
    a[i] = 0;
    b[i] = i;
    c[i] = 1;
  }

  double *d_a, *d_b, *d_c;
  int e = 0;
  e = hipMalloc(&d_a, SIZE*sizeof(double));
  e |= hipMalloc(&d_b, SIZE*sizeof(double));
  e |= hipMalloc(&d_c, SIZE*sizeof(double));




  e |= hipMemcpy(d_a, a, SIZE*sizeof(double), hipMemcpyHostToDevice);
  e |= hipMemcpy(d_b, b, SIZE*sizeof(double), hipMemcpyHostToDevice);
  e |= hipMemcpy(d_c, c, SIZE*sizeof(double), hipMemcpyHostToDevice);



  auto time_start = high_resolution_clock::now();
  vector_add<<<1024,1024>>>(d_a, d_b, d_c, SIZE);
  e |= hipDeviceSynchronize();

  auto time_end = high_resolution_clock::now();
  auto time_duration = duration_cast<nanoseconds>(time_end - time_start);
  double time_seconds = time_duration.count()/1000000000.0;
  
  cout << "timings: " << time_seconds << endl;

  hipMemcpy(a, d_a, SIZE*sizeof(double), hipMemcpyDeviceToHost); 
      
  for (int i = 0; i < SIZE; i++) {
    assert(a[i] == (i + 1));    
  }  

  return 0;
}


  
